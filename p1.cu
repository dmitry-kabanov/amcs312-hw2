#include "hip/hip_runtime.h"
#include <stdio.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("Memory clock rate: %d kHz\n", prop.memoryClockRate);
    printf("Memory bus width: %d bits\n", prop.memoryBusWidth);

    return 0;
}
