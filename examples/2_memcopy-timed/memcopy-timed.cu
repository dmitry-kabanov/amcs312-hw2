#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// CPU code
int main(int argc, char* argv[])
{
	if(argc < 2)
	{
		printf("USGAE: %s <array-length>\n", argv[0]);
		exit(-1);
	}
	
	int length = atoi(argv[1]);
	
	// cpu pointers
	float *ha, *hb;
	// gpu pointers
	float *da, *db;
	
	// cuda events for timing
	hipEvent_t start, stop;
	
	//elapsed times
	float time_h2d, time_d2d, time_d2h; 
	
	// create cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// allocate and initialize cpu memory
	ha = (float*)malloc(length * sizeof(float));
	hb = (float*)malloc(length * sizeof(float));
	for(int i = 0; i < length; i++) ha[i] = rand();		// init ha randomly
	memset(hb, 0, length * sizeof(float));				// init hb with zeros
	
	// allocate gpu memory
	hipMalloc((void**)&da, length * sizeof(float));
	hipMalloc((void**)&db, length * sizeof(float));
	
	printf("Copying from host to device .. ");
	hipEventRecord(start, 0);
	// memory copy host to device
	hipMemcpy(da, ha, length * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_h2d, start, stop);
	printf("done\n\n");
	
	printf("Copying inside device memory .. ");
	hipEventRecord(start, 0);
	// memory copy inside gpu memory
	hipMemcpy(db, da, length * sizeof(float), hipMemcpyDeviceToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_d2d, start, stop);
	printf("done\n\n");
	
	printf("Copying back from device to host .. ");
	hipEventRecord(start, 0);
	// memory copy from device (gpu) to host (cpu)
	hipMemcpy(hb, db, length * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_d2h, start, stop);
	printf("done\n\n");
	
	
	printf("Checking gpu output .. ");
	
	// now ha and hb should be the same
	int cmp = memcmp(ha, hb, length * sizeof(float) );
	if(cmp == 0)printf("passed\n");
	else printf("failed at %d\n", cmp);
	
	printf("\n------------------------------------------\n");
	printf("Copy time (host   to device): %.2f ms\n", time_h2d);
	printf("Copy time (device to device): %.2f ms\n", time_d2d);
	printf("Copy time (device to host  ): %.2f ms\n", time_d2h);
	printf("------------------------------------------\n");
	// destroy cuda events
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	// free cpu memory
	if(ha)free(ha);
	if(hb)free(hb);
	
	// free gpu memory
	if(da)hipFree(da);
	if(db)hipFree(db);
	return 0;	
}