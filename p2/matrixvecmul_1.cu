#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__
void matrixmul_1(float* A, float* x, float* y, int dim, int block_size)
{
	const int row = blockIdx.y * block_size + threadIdx.y;
	
	float sum = 0.0;
	for(int k = 0; k < dim; k++)
		sum += A[row * dim + k] * x[k];
	
	y[row] = sum;
}

extern "C"
void matrixmul_1_driver(float* dA, float* dx, float* dy, int dim, int block_size)
{
	if(dim % block_size != 0){printf("ERROR: Block size does not fully divide matrix dimension .. exiting\n"); return;}
	
	// thread block configuration
	int block_dim_x = block_size;
	int block_dim_y = block_size;
	
	// kernel grid configuration 
	int grid_dim_x = dim / block_size;
	int grid_dim_y = dim / block_size;

	dim3 dimBlock(block_dim_x, block_dim_y);
	dim3 dimGrid(grid_dim_x, grid_dim_y);
	
	matrixmul_1<<<dimGrid, dimBlock>>>(dA, dx, dy, dim, block_size);
}
